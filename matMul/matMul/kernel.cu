#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cmath>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 16
#define TILE_SIZE 16

hipError_t multWithCuda(
	double* c, 
	const double* a, 
	const double* b, 
	int hc, int wc,
	int ha, int wa,
	int hb, int wb);

void init(double* data, int size);

__global__ void
matrixMul(
	double* c,
	double* a,
	double* b, 
	int hc, int wc,
	int ha, int wa, 
	int hb, int wb)
{
   int tx = blockIdx.x * TILE_SIZE + threadIdx.x;
   int ty = blockIdx.y * TILE_SIZE + threadIdx.y;
 
   double value = 0;
   for (int i = 0; i < wa; ++i)
   {
      double elementA = a[ty * wa + i];
      double elementB = b[i * wb + tx];
      value += elementA * elementB;
   }
 
   c[ty * wa + tx] = value;
}

void debug_print(double* data, int size, int rowSize)
{
	printf("\n==================================================\n");
    for(int i = 0; i < size; i++)
    {
       printf("%f ", data[i]);
       if(((i + 1) % rowSize) == 0)
          printf("\n");
    }
}

bool check_result(double *c, double *a, double *b, int hc, int wc, int ha, int wa, int hb, int wb)
{
	bool isCorrect = true;
	double eps = 1e-6;

	for(int i = 0; i < ha; i++) //ha = wb
	{
		for(int j = 0; j < wb; j++) 
		{
			double cur = 0;

			for(int k = 0; k < wa; k++) //wa = hb
				cur += a[i * wa + k] *  b[k * wb + j];
			if(fabs(cur - c[i * wc + j]) > eps)
				isCorrect = false;
		}
	}
		
	return isCorrect;
}

int main(int argc, char *argv[])
{
	//int n = argc == 1 ? 2048 : atoi(argv[1]);
	int n = 1024;
	int ha = n;
	int wa = n;
	int hb = n;
	int wb = n;
	int hc = n;
	int wc = n;

 
    int sizeA = ha * wa;
    int sizeB = hb * wb;
    int sizeC = hc * wc;

    double* a = (double*) malloc(sizeof(double) * sizeA);
    double* b = (double*) malloc(sizeof(double) * sizeB);
    double* c = (double*) malloc(sizeof(double) * sizeC);
 
	srand(146);
    init(a, sizeA);
    init(b, sizeB);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time = 0;

	hipEventRecord(start, 0);

	hipError_t cudaStatus = multWithCuda(c, a, b,
		hc, wc,
		ha, wa,
		hb, wb);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	bool isSuccess = check_result(c, a, b, hc, wc, ha, wa, hb, wb);
	printf("%dKB. %s. ElapsedTime is %.6f\n", sizeof(double) * sizeC / 1024, isSuccess ? "Success" : "Fail", time);


    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	//debug_print(c, sizeC, wc);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	free(a);
    free(b);
    free(c);

    return 0;
}

void init(double* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (double)RAND_MAX;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multWithCuda(
	double *c, 
	const double *a, 
	const double *b, 
	int hc, int wc,
	int ha, int wa,
	int hb, int wb)
{
    double *dev_a = 0;
    double *dev_b = 0;
    double *dev_c = 0;
    hipError_t cudaStatus;

	int sizeC = hc * wc;
	int sizeA = ha * wa;
	int sizeB = hb * wb;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, sizeC * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, sizeA * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, sizeB * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, sizeA * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, sizeB * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(wc / threads.x, hc / threads.y);
 
    // Launch a kernel on the GPU with one thread for each element.
    matrixMul<<< grid, threads >>>(dev_c, dev_a, dev_b, 
		hc, wc,
		ha, wa,
		hb, wb);
 
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, sizeC * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

	return cudaStatus;
}