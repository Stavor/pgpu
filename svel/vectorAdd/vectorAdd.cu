#include <stdio.h>
#include <hip/hip_runtime.h>

static int threadsPerBlock = 256;

__global__ void reductionByAdd(int *a, int *res, int elCnt) {
	int firstThreadBlockId = blockIdx.x * blockDim.x;	
	int threadId = firstThreadBlockId + threadIdx.x;

	for(int i = 1; i < blockDim.x; i <<= 1) {
		if(threadId % (2 * i) == 0 && threadId < elCnt && threadId + i < elCnt) {
			a[threadId] += a[threadId + i];
		}
		__syncthreads();
	}
	if(threadId == firstThreadBlockId)
		res[blockIdx.x] = a[firstThreadBlockId];
}

void checkCudaError(hipError_t error) {
	if(error != hipSuccess)
	{
		fprintf(stderr, "Failed! (error cdoe %s)\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

__host__ void reductionAdd(int *a, int *res, int elCnt) {
	int blocksPerGrid = (elCnt + threadsPerBlock - 1) / threadsPerBlock;
	size_t sizeRes = blocksPerGrid * sizeof(int);
	
	int *result = NULL;
	hipError_t error = hipMalloc((void**)&result, sizeRes);
	checkCudaError(error);

	reductionByAdd<<<blocksPerGrid, threadsPerBlock>>>(a, result, elCnt);
	if(elCnt > threadsPerBlock) {
		reductionAdd(result, res, blocksPerGrid);
		checkCudaError(error);
	} else {
		error = hipMemcpy(res, result, sizeof(int), hipMemcpyDeviceToHost);
		checkCudaError(error);
	}

	error =  hipFree(result);
	checkCudaError(error);
}

int main() {
	int elementCnt = 7000000;

	size_t size1 = elementCnt * sizeof(int);
	
	int *hosta = (int *)malloc(size1);
	int *ans = (int *)malloc(sizeof(int));

	if(hosta == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	for(int i = 0; i < elementCnt; ++i) {
		hosta[i] = rand() % 100;
	}

	int *deva1 = NULL;
	hipError_t error = hipMalloc((void**)&deva1, size1);
	checkCudaError(error);

	error = hipMemcpy(deva1, hosta, size1, hipMemcpyHostToDevice);
	checkCudaError(error);

    reductionAdd(deva1, ans, elementCnt);

    error = hipGetLastError();
    checkCudaError(error);

    //Check
    int res = 0;
    for (int i = 0; i < elementCnt; ++i) {
    	res += hosta[i];
    }
    printf("%d\n", res);
    printf("%d\n", *ans);
    printf("%d\n", abs(res - *ans));

    if (abs(res - *ans) != 0) {
            fprintf(stderr, "Result verification failed!\n");
            exit(EXIT_FAILURE);
    }

    error = hipFree(deva1);
	checkCudaError(error);

    free(hosta);
    free(ans);

    error = hipDeviceReset();
	checkCudaError(error);

    printf("Done\n");
    return 0;
}